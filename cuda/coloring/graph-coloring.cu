
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>
#include <unordered_map>

bool verbose = true;

std::vector<std::vector<uint64_t>> readGraph (const std::string& path)
{
	// Open the file and make sure it exists
	std::ifstream remapStream;
	remapStream.open(path.c_str());

	// Remap the vertex ids
	if (verbose) std::cout << "\tRemapping vertex IDs...\n";

	std::vector<std::string> lines;
	std::string line;

	uint64_t a, b;
	uint64_t highestId = 0;

	std::unordered_map<uint64_t, uint64_t> vertexIdMap; // <original, remapped>
	
	while (std::getline(remapStream, line))
	{
		if (line[0] == '%') 
		{
			if (verbose) std::cout << "\t\tSkip" << std::endl;
			continue;
		}

		std::istringstream string(line);
		string >> a >> b;
		
		auto search = vertexIdMap.find(a);
		if (search == vertexIdMap.end())
		{
			vertexIdMap.insert(std::make_pair(a, highestId++));
			if (verbose) std::cout << "\t\tMap " << a << " to " << highestId << std::endl;
		}
		
		search = vertexIdMap.find(b);
		if (search == vertexIdMap.end())
		{
			vertexIdMap.insert(std::make_pair(b, highestId++));
			if (verbose) std::cout << "\t\tMap " << b << " to " << highestId << std::endl;
		}
	}

	if (verbose) std::cout << "\tDone\n";

	// Create adjacency list
	if (verbose) std::cout << "\tCreating adjacency list...\n";

	std::vector<std::vector<uint64_t>> graph(vertexIdMap.size());
	
	std::ifstream neighborStream;
	neighborStream.open(path.c_str());

	while (std::getline(neighborStream, line))
	{
		if (line[0] == '%')
		{
			if (verbose) std::cout << "\t\tSkip\n";
			continue;
		}

		std::istringstream string(line);
		string >> a >> b;

		if (a != b)
		{
			uint64_t remappedA = vertexIdMap.find(a)->second;
			uint64_t remappedB = vertexIdMap.find(b)->second;

			graph.at(remappedA).push_back(remappedB);
			graph.at(remappedB).push_back(remappedA);

			if (verbose) std::cout << "\t\tAdd edge between "
				<< "vertex " << remappedA << " (originally " << a << ") and "
				<< "vertex " << remappedB << " (originally " << b << ")\n";
		}
	}

	if (verbose) std::cout << "\tDone\n";

	return graph;
}

int main (int argc, char *argv[])
{

	// Make sure a graph is passed
	if (argc != 2)
	{
		std::cout << "No graph provided!\n";
		return 1;
	}

	// Read the graph
	if (verbose) std::cout << "Reading graph...\n";
	std::string path(argv[1]);
	std::vector<std::vector<uint64_t>> lines = readGraph(path);
	if (verbose) std::cout << "Done\n";

	return 0;
}
