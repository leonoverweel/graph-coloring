
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>
#include <unordered_map>
#include <set>
#include <random>
#include <algorithm>
#include <iterator>
#include <functional>

bool verbose = true;

// Read a graph from a file
std::vector<std::vector<uint64_t>> readGraph (const std::string& path)
{
	// Open the file and make sure it exists
	std::ifstream remapStream;
	remapStream.open(path.c_str());

	// Remap the vertex ids
	if (verbose) std::cout << "\tRemapping vertex IDs...\n";

	std::vector<std::string> lines;
	std::string line;

	uint64_t a, b;
	uint64_t highestId = 0;

	std::unordered_map<uint64_t, uint64_t> vertexIdMap; // <original, remapped>
	
	while (std::getline(remapStream, line))
	{
		if (line[0] == '%') 
		{
			if (verbose) std::cout << "\t\tSkip" << std::endl;
			continue;
		}

		std::istringstream string(line);
		string >> a >> b;
		
		auto search = vertexIdMap.find(a);
		if (search == vertexIdMap.end())
		{
			vertexIdMap.insert(std::make_pair(a, highestId));
			if (verbose) std::cout << "\t\tMap " << a << " to " << highestId << std::endl;
			highestId++;
		}
		
		search = vertexIdMap.find(b);
		if (search == vertexIdMap.end())
		{
			vertexIdMap.insert(std::make_pair(b, highestId));
			if (verbose) std::cout << "\t\tMap " << b << " to " << highestId << std::endl;
			highestId++;
		}
	}

	if (verbose) std::cout << "\tDone\n";

	// Create adjacency list
	if (verbose) std::cout << "\tCreating adjacency list...\n";

	std::vector<std::vector<uint64_t>> graph(vertexIdMap.size());
	
	std::ifstream neighborStream;
	neighborStream.open(path.c_str());

	while (std::getline(neighborStream, line))
	{
		if (line[0] == '%')
		{
			if (verbose) std::cout << "\t\tSkip\n";
			continue;
		}

		std::istringstream string(line);
		string >> a >> b;

		if (a != b)
		{
			uint64_t remappedA = vertexIdMap.find(a)->second;
			uint64_t remappedB = vertexIdMap.find(b)->second;

			graph.at(remappedA).push_back(remappedB);
			graph.at(remappedB).push_back(remappedA);

			if (verbose) std::cout << "\t\tAdd edge between "
				<< "vertex " << remappedA << " (originally " << a << ") and "
				<< "vertex " << remappedB << " (originally " << b << ")\n";
		}
	}

	if (verbose) std::cout << "\tDone\n";

	// Sort neighbor lists and remove duplicates
	if (verbose) std::cout << "\tSorting neighbor lists and removing duplicates... ";

	for (int i = 0; i < graph.size(); i++)
	{
		std::vector<uint64_t> * neighborList = &graph[i];
		std::set<uint64_t> set(neighborList->begin(), neighborList->end());
		neighborList->assign(set.begin(), set.end());	
	}

	if (verbose) std::cout << "Done\n";

	// Print the final adjacency list
	if (verbose)
	{
		std::cout << "\tFinal adjacency list:\n";
		for(int i = 0; i < graph.size(); i++)
		{
			std::cout << "\t\t" << i << ": ";
			for(uint64_t neighbor : graph.at(i))
			{
				std::cout << neighbor << ", ";
			}
			std::cout << "\b\b \n";
		}
	}

	return graph;
}

// Flatten a graph into a single data vector with pointers to where each vertex's neighbor list starts and ends. Returns size in bytes.
int flatten (std::vector<std::vector<uint64_t>> * graph, std::vector<uint64_t> * data, std::vector<uint64_t> * indices)
{
	indices->push_back(0);

	for (int i = 0; i < graph->size(); i++)
	{
		data->insert(data->end(), (*graph)[i].begin(), (*graph)[i].end());
		indices->push_back(data->size());
	}

	if (verbose) std::cout << "\tLength of data: " << data->size() << std::endl;
	if (verbose) std::cout << "\tSize in bytes: " << data->size() * sizeof(uint64_t) << std::endl;

	if (verbose)
	{
		std::cout << "\tAdjacency list from flattened data:\n";
		for(int i = 0; i < indices->size() - 1; i++)
		{
			std::cout << "\t\t" << i << ": ";
			for(uint64_t j = (*indices)[i]; j < (*indices)[i+1]; j++) 
		{
				std::cout << (*data)[j] << ", ";
			}
			std::cout << "\b\b \n";
		}
	}

	return data->size() * sizeof(uint64_t);
}

int main (int argc, char *argv[])
{

	// Make sure a graph is passed
	if (argc != 2)
	{
		std::cout << "No graph provided!\n";
		return 1;
	}

	// Read the graph
	if (verbose) std::cout << "Reading graph...\n";

	std::string path(argv[1]);
	std::vector<std::vector<uint64_t>> graph = readGraph(path);

	if (verbose) std::cout << "Done\n";
	
	// Generate the random numbers
	if (verbose) std::cout << "Generating random numbers...\n";

	std::random_device rnd_device;
	std::mt19937 mersenne_engine(rnd_device());
	std::uniform_int_distribution<int> dist(INT_MIN, INT_MAX);
	auto gen = std::bind(dist, mersenne_engine);

	std::vector<int> * random = new std::vector<int>(graph.size());
	int randomSize = random->size() * sizeof(int);

	if (verbose) std::cout << "\tSize: " << randomSize << std::endl;

	std::generate(random->begin(), random->end(), gen);

	if (verbose)
	{
		for (int i = 0; i < graph.size(); i++)
		{
			std::cout << "\t" << i << ": " << random->at(i) << std::endl;
		}
	}

	if (verbose) std::cout << "Done\n";

	// Flatten the graph
	if (verbose) std::cout << "Flattening the graph...\n";
	
	std::vector<uint64_t> * data = new std::vector<uint64_t>();
	std::vector<uint64_t> * indices = new std::vector<uint64_t>();

	int dataSize = flatten(&graph, data, indices);
	int indicesSize = indices->size() * sizeof(uint64_t);

	if (verbose) std::cout << "Done\n";

	// Send data to device
	if (verbose) std::cout << "Sending graph to device...";
	
	uint64_t * deviceDataPointer;
	uint64_t * deviceIndicesPointer;
	uint64_t * deviceRandomPointer;

	hipMalloc((void**)&deviceDataPointer, dataSize);
	hipMalloc((void**)&deviceIndicesPointer, indicesSize);
	hipMalloc((void**)&deviceRandomPointer, randomSize);

	hipMemcpy(deviceDataPointer, data, dataSize, hipMemcpyHostToDevice);
	hipMemcpy(deviceIndicesPointer, indices, indicesSize, hipMemcpyHostToDevice);
	hipMemcpy(deviceRandomPointer, random, randomSize, hipMemcpyHostToDevice);

	if (verbose) std::cout << "Done\n";

	return 0;
}
