
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>
#include <unordered_map>

std::vector<std::vector<uint64_t>> readGraph (const std::string& path)
{
	std::vector<std::vector<uint64_t>> graph;

	// Open the file and make sure it exists
	std::ifstream remapStream;
	remapStream.open(path.c_str());

	if (!remapStream) return graph;

	// Remap the vertex ids
	std::vector<std::string> lines;
	std::string line;

	uint64_t a, b;
	uint64_t highestId = 0;

	std::unordered_map<uint64_t, uint64_t> vertexIdMap; // <original, remapped>
	
	while (std::getline(remapStream, line))
	{
		if (line[0] == '%') 
		{
			std::cout << "Skipped" << std::endl;
			continue;
		}

		std::istringstream string(line);
		string >> a >> b;
		
		auto search = vertexIdMap.find(a);
		if (search == vertexIdMap.end())
		{
			vertexIdMap.insert(std::make_pair(a, highestId++));
			std::cout << "Map " << a << " to " << highestId << std::endl;
		}
		
		search = vertexIdMap.find(b);
		if (search == vertexIdMap.end())
		{
			vertexIdMap.insert(std::make_pair(b, highestId++));
			std::cout << "Map " << b << " to " << highestId << std::endl;
		}
	}

	return graph;
}

int main (int argc, char *argv[])
{

	// Make sure a graph is passed
	if (argc != 2)
	{
		std::cout << "No graph provided!\n";
		return 1;
	}

	// Read the graph
	std::string path(argv[1]);
	std::vector<std::vector<uint64_t>> lines = readGraph(path);

	return 0;
}
