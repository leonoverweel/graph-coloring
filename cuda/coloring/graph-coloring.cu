
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

std::vector<std::string> readFile( const std::string& path )
{
	// Open the file and make sure it exists
	std::ifstream source;
	source.open(path.c_str());

	// Iterate over the file's lines
	std::vector<std::string> lines;
	std::string line;

	if( !source ) return lines;

	while( std::getline(source, line) )
	{
		std::cout << line << std::endl;
		lines.push_back(line);
	}

	return lines;
}

int main( int argc, char *argv[] )
{

	// Make sure a graph is passed
	if( argc != 2)
	{
		std::cout << "No graph provided!\n";
		return 1;
	}

	// Read the graph
	std::string path(argv[1]);
	std::vector<std::string> lines = readFile(path);

	return 0;
}
