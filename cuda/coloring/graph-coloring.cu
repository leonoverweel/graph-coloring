
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>
#include <unordered_map>

bool verbose = true;

std::vector<std::vector<uint64_t>> readGraph (const std::string& path)
{
	std::vector<std::vector<uint64_t>> graph;

	// Open the file and make sure it exists
	std::ifstream remapStream;
	remapStream.open(path.c_str());

	if (!remapStream) return graph;

	// Remap the vertex ids
	std::vector<std::string> lines;
	std::string line;

	uint64_t a, b;
	uint64_t highestId = 0;

	std::unordered_map<uint64_t, uint64_t> vertexIdMap; // <original, remapped>
	
	while (std::getline(remapStream, line))
	{
		if (line[0] == '%') 
		{
			if (verbose) std::cout << "\tSkipped" << std::endl;
			continue;
		}

		std::istringstream string(line);
		string >> a >> b;
		
		auto search = vertexIdMap.find(a);
		if (search == vertexIdMap.end())
		{
			vertexIdMap.insert(std::make_pair(a, highestId++));
			if (verbose) std::cout << "\tMap " << a << " to " << highestId << std::endl;
		}
		
		search = vertexIdMap.find(b);
		if (search == vertexIdMap.end())
		{
			vertexIdMap.insert(std::make_pair(b, highestId++));
			if (verbose) std::cout << "\tMap " << b << " to " << highestId << std::endl;
		}
	}

	return graph;
}

int main (int argc, char *argv[])
{

	// Make sure a graph is passed
	if (argc != 2)
	{
		std::cout << "No graph provided!\n";
		return 1;
	}

	// Read the graph
	if (verbose) std::cout << "Reading graph...\n";
	std::string path(argv[1]);
	std::vector<std::vector<uint64_t>> lines = readGraph(path);
	if (verbose) std::cout << "Done\n";

	return 0;
}
