
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

bool verbose = true;
int SIZE = 10;

__global__
void assign_state(uint8_t * states)
{
	states[threadIdx.x] = 42;
}

int main (int argc, char *argv[])
{

	// Create host states vector
	if (verbose) std::cout << "Creating host memory... ";

	std::vector<uint8_t> * states = new std::vector<uint8_t>(SIZE);
	int statesSize = states->size() * sizeof(uint8_t);
	
	if (verbose) std::cout << "Done\n";

	// Send data to device
	if (verbose) std::cout << "Sending graph to device... ";
	
	uint8_t * deviceStatesPointer;
	hipMalloc((void**)&deviceStatesPointer, statesSize);
	hipMemcpy(deviceStatesPointer, states, statesSize, hipMemcpyHostToDevice);

	if (verbose) std::cout << "Done\n";

	// Assign states
	if (verbose) std::cout << "Assign states... ";

	dim3 dimBlock(SIZE, 1);
	dim3 dimGrid(1, 1);
	assign_state<<<dimGrid, dimBlock>>>(deviceStatesPointer);

	if (verbose) std::cout << "Done\n";

	// Get data back from device
	if (verbose) std::cout << "Getting data back from device...\n";

	hipMemcpy(states, deviceStatesPointer, statesSize, hipMemcpyDeviceToHost);

	if (verbose)
	{
		for (int i = 0; i < states->size(); i++)
		{
			std::cout << "\t" << i << ": " << states->at(i) << std::endl;

		}
	}

	if (verbose) std::cout << "Done\n";

	return 0;
}
